#include<hip/hip_runtime.h>
__global__ void add2_kernel(float* c,
                            const float* a,
                            const float* b,
                            int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
            i < n; i += gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
    }
}

void launch_add2(float* c,
                 const float* a,
                 const float* b,
                 int n) {
    dim3 grid((n + 1023) / 1024);
    dim3 block(1024);
    add2_kernel<<<grid, block>>>(c, a, b, n);
}

// __global__ void add2_kernel(float* c,
//                             const float* a,
//                             const float* b,
//                             int n) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     if(i<n) {
//         c[i] = a[i] + b[i];
//     }
// }

// void launch_add2(float* c,
//                  const float* a,
//                  const float* b,
//                  int n) {


//     dim3 grid((n + 1023) / 1024);
//     dim3 block(1024);
//     int N = n;

//     float *d_a, *d_b, *d_c;
//     cudaMalloc((void**)&d_a, sizeof(float)*N);
//     cudaMalloc((void**)&d_b, sizeof(float)*N);
//     cudaMalloc((void**)&d_c, sizeof(float)*N);

//     cudaMemcpy(d_a, a, sizeof(float)*N, cudaMemcpyHostToDevice);
//     cudaMemcpy(d_b, b, sizeof(float)*N, cudaMemcpyHostToDevice);
//     add2_kernel<<<1, N>>>(d_c, d_a, d_b, n);
//     // cudaDeviceSynchronize();
//     cudaMemcpy(c, d_c, sizeof(float)*N, cudaMemcpyDeviceToHost);
//     cudaFree(d_a);
//     cudaFree(d_b);
//     cudaFree(d_c);
// }